#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#define N 1000000
#define THREADS_PER_BLOCK 256

__device__ void collatz(uint64_t x, uint64_t* steps, uint64_t* peak) {
    uint64_t n = x;
    *steps = 0;
    *peak = x;
    while (n != 1) {
        if (n & 1) {
            n = 3 * n + 1;
        } else {
            n >>= 1;
        }
        if (n > *peak) *peak = n;
        (*steps)++;
    }
}

__global__ void collatz_kernel(uint64_t* steps_arr, uint64_t* peak_arr) {
    uint64_t idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (idx <= N) {
        uint64_t steps, peak;
        collatz(idx, &steps, &peak);
        steps_arr[idx - 1] = steps;
        peak_arr[idx - 1] = peak;
    }
}

int main() {
    printf("hello world!\n");

    uint64_t *steps_arr, *peak_arr;
    uint64_t *d_steps_arr, *d_peak_arr;
    size_t size = N * sizeof(uint64_t);

    steps_arr = (uint64_t*)malloc(size);
    peak_arr = (uint64_t*)malloc(size);
    hipError_t err;

    err = hipMalloc(&d_steps_arr, size);
    if (err != hipSuccess) { printf("hipMalloc d_steps_arr error: %s\n", hipGetErrorString(err)); return 1; }
    err = hipMalloc(&d_peak_arr, size);
    if (err != hipSuccess) { printf("hipMalloc d_peak_arr error: %s\n", hipGetErrorString(err)); return 1; }

    int blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    collatz_kernel<<<blocks, THREADS_PER_BLOCK>>>(d_steps_arr, d_peak_arr);
    err = hipGetLastError();
    if (err != hipSuccess) { printf("Kernel launch error: %s\n", hipGetErrorString(err)); return 1; }
    err = hipDeviceSynchronize();
    if (err != hipSuccess) { printf("hipDeviceSynchronize error: %s\n", hipGetErrorString(err)); return 1; }

    err = hipMemcpy(steps_arr, d_steps_arr, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { printf("hipMemcpy steps_arr error: %s\n", hipGetErrorString(err)); return 1; }
    err = hipMemcpy(peak_arr, d_peak_arr, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) { printf("hipMemcpy peak_arr error: %s\n", hipGetErrorString(err)); return 1; }

    // Reduction on CPU for simplicity
    uint64_t best_n = 1, best_steps = 0, best_peak = 1, xor_steps = 0;
    for (uint64_t i = 0; i < N; ++i) {
        xor_steps ^= steps_arr[i];
        if (steps_arr[i] > best_steps) {
            best_steps = steps_arr[i];
            best_n = i + 1;
            best_peak = peak_arr[i];
        }
    }

    printf("collatz_longest(1..%d)\n", N);
    printf("n*=%llu\n", (unsigned long long)best_n);
    printf("steps=%llu\n", (unsigned long long)best_steps);
    printf("peak=%llu\n", (unsigned long long)best_peak);
    printf("xor_steps=%llu\n", (unsigned long long)xor_steps);

    hipFree(d_steps_arr);
    hipFree(d_peak_arr);
    free(steps_arr);
    free(peak_arr);
    return 0;
}
